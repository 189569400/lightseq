#include "hip/hip_runtime.h"
#include "t5Kernels.h"
#include "common.h"

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

/**
@file
Implemented the cuda kernel function and its launcher
that required by transformer model.
Currently, fp16 and fp32 versions are provided
*/
namespace lightseq {
namespace cuda {
  /**
  @brief: t5_ker_norm_layer
  layer normalization

  @thread
  gridDim.x = batch_size * batch_seq_len
  blockDim.x = max_thread_per_block

  @param
  matrix: [batch_size, batch_seq_len, hidden_size]
  scale: [hidden_size]
  bias: [hidden_size]
  */
  template <typename T>
  __global__ void t5_ker_norm_layer(T* matrix, T* out, const T* scale, const T* bias,
                                int hidden_size) {
    uint block_start = blockIdx.x * hidden_size;
    uint start = block_start + threadIdx.x;
    uint end = block_start + hidden_size;
    // for (uint i = start; i < end; i += blockDim.x) {
    //   val += matrix[i];
    // }
    // step 0. compute mean
    // __shared__ float s_mean;
    // float reduce_res = blockReduceSum<float>(val);
    // if (threadIdx.x == 0) s_mean = reduce_res / float(hidden_size);
    // __syncthreads();
    
    float val = 0.0;
    // step 1. compute variance
    for (uint i = start; i < end; i += blockDim.x) {
      float tmp = matrix[i]; // - s_mean;
      val += tmp * tmp;
    }
    __shared__ float s_var;
    float reduce_res = blockReduceSum(val);
    if (threadIdx.x == 0) {
      s_var = rsqrtf(reduce_res / float(hidden_size) + t5_epsilon);
    }
    __syncthreads();


    // step 2. layer norm
    for (uint i = start; i < end; i += blockDim.x) {
      // val = matrix[i] - s_mean;
      out[i] = matrix[i] * s_var * __ldg(&scale[i - block_start]) +
                  __ldg(&bias[i - block_start]);
    }
  }


  template <>
  __global__ void t5_ker_norm_layer<__half>(__half* matrix, __half* out, const __half* scale,
                                        const __half* bias,
                                        int half_hidden_size) {
    uint block_start = blockIdx.x * half_hidden_size;
    uint start = block_start + threadIdx.x;
    uint end = blockIdx.x * half_hidden_size + half_hidden_size;
    half2* pmatrix = (half2*)matrix;
    half2* pout = (half2*)out;
    const half2* pscale = (const half2*)scale;
    const half2* pbias = (const half2*)bias;
    float mean_dim = float(half_hidden_size) * 2.f;

    float val = 0.0;
    // step 0. compute mean
    // for (uint i = start; i < end; i += blockDim.x) {
    //   float2 local_f2 = safe_half2_to_float2(pmatrix[i]);
    //   val += local_f2.x + local_f2.y;
    // }
    // __shared__ float s_mean;
    // float reduce_res = blockReduceSum<float>(val);
    // if (threadIdx.x == 0) s_mean = reduce_res / mean_dim;
    // __syncthreads();

    // step 1. compute variance
    val = 0.0;
    for (uint i = start; i < end; i += blockDim.x) {
      float2 local_f2 = safe_half2_to_float2(pmatrix[i]);
      // float tmpx = local_f2.x - s_mean;
      // float tmpy = local_f2.y - s_mean;
      float tmpx = local_f2.x;
      float tmpy = local_f2.y;
      val += tmpx * tmpx + tmpy * tmpy;
    }
    __shared__ float s_var;
    float reduce_res = blockReduceSum(val);
    if (threadIdx.x == 0)
      s_var = rsqrtf(reduce_res / mean_dim + t5_epsilon);

    __syncthreads();

    // step 2. layer norm
    for (uint i = start; i < end; i += blockDim.x) {
      float2 scale_val = __half22float2(__ldg(&pscale[i - block_start]));
      float2 bias_val = __half22float2(__ldg(&pbias[i - block_start]));
      float2 local_f2 = safe_half2_to_float2(pmatrix[i]);
      // local_f2.x = (local_f2.x - s_mean) * s_var * scale_val.x + bias_val.x;
      // local_f2.y = (local_f2.y - s_mean) * s_var * scale_val.y + bias_val.y;
      local_f2.x = local_f2.x * s_var * scale_val.x + bias_val.x;
      local_f2.y = local_f2.y * s_var * scale_val.y + bias_val.y;
      pout[i] = __float22half2_rn(local_f2);
    }
  }

  template <typename T>
  void t5_ker_norm_layer_launcher(int token_num, int hidden_size,
                              hipStream_t stream, T* matrix, T* out, const T* scale,
                              const T* bias, int max_thread_per_block) {
    t5_ker_norm_layer<T><<<token_num, max_thread_per_block, 0, stream>>>(
        matrix, out, scale, bias, hidden_size);
  }

  template <>
  void t5_ker_norm_layer_launcher<__half>(int token_num, int hidden_size,
                                      hipStream_t stream, __half* matrix, __half* out,
                                      const __half* scale, const __half* bias,
                                      int max_thread_per_block) {
    t5_ker_norm_layer<__half><<<token_num, max_thread_per_block, 0, stream>>>(
        matrix, out, scale, bias, hidden_size / 2);
  }


  __device__ int get_bucket_num(int row, int col, bool bidirectional, int num_buckets=32, int max_distance=128) {
    int relative_position = col - row;
    int relative_buckets = 0;
    if (bidirectional) {
        num_buckets /= 2;
        if (relative_position > 0) relative_buckets += num_buckets;
        relative_position = abs(relative_position);
    } else
        relative_position = -min(relative_position, 0);

    int max_exact = num_buckets / 2;
    int relative_position_if_large = max_exact + (
        log((double)relative_position / max_exact)
        / log((double)max_distance / max_exact)
        * (num_buckets - max_exact)
    );

    if (relative_position < max_exact)
        relative_buckets += relative_position;
    else
        relative_buckets += relative_position_if_large;
    return relative_buckets;
  }

  /**
  @brief: t5_ker_correlation_softmax_encself
  query-key correlation softmax for encoder self attention

  @thread
  gridDim.x = batch_size
  gridDim.y = head_num * batch_seq_len
  blockDim.x = first multiple of WARP_SIZE greater than batch_seq_len

  @param
  correlation: [batch_size, head_num, batch_seq_len, batch_seq_len]
  src_padding_mask: [batch_size, batch_seq_len],
    indicating which token is a padding token.
  */
  template <typename T>
  __global__ void t5_ker_correlation_softmax_encself(T* correlation,
                                                  const int* src_padding_mask,
                                                  int batch_seq_len,
                                                  const T *pos_emb) {
    int idx = (blockIdx.x * gridDim.y + blockIdx.y) * batch_seq_len + threadIdx.x;
    if (threadIdx.x < batch_seq_len &&
        src_padding_mask[blockIdx.x * batch_seq_len +
                          blockIdx.y % batch_seq_len]) {
      correlation[idx] = (T)0.f;
      return;
    }
    int mask = threadIdx.x < batch_seq_len
                    ? src_padding_mask[blockIdx.x * batch_seq_len + threadIdx.x]
                    : 1;
    // float val = threadIdx.x < batch_seq_len ? (float)correlation[idx]
    //                                         : CUDA_FLOAT_INF_NEG;
    float val;
    if (threadIdx.x < batch_seq_len) {
      // We know that idx = head_num * batch_seq_len * batch_seq_len
      //     + i * batch_seq_len + j;
      int j = idx % batch_seq_len;
      int i = (idx - j) / batch_seq_len % batch_seq_len;
      int head_idx = (idx - j - i * batch_seq_len) / batch_seq_len / batch_seq_len;
      val = (float)correlation[idx];
      // new_values[0, head, i, j] = relative_attention_bias.weight[relative_position_bucket[i][j]][head]
      int bucket_index = get_bucket_num(i, j, true);
      val += (float)pos_emb[bucket_index * 8 + head_idx];
    } else val = CUDA_FLOAT_INF_NEG;

    float max_val = blockReduceMax<float>(mask ? CUDA_FLOAT_INF_NEG : val);
    __shared__ float smax;
    if (threadIdx.x == 0) smax = max_val;
    __syncthreads();

    val = mask ? 0.f : expf(val - smax);
    float rsum = blockReduceSum<float>(val);
    __shared__ float ssum;
    if (threadIdx.x == 0) ssum = rsum;
    __syncthreads();

    if (threadIdx.x < batch_seq_len) correlation[idx] = (T)(val / ssum);
  }


  template <typename T>
  void t5_ker_correlation_softmax_encself_launcher(int batch_size, int batch_seq_len,
                                                int head_num, hipStream_t stream,
                                                T* correlation,
                                                const int* src_padding_mask,
                                                const T *pos_emb) {
    int block_dim = batch_seq_len;
    if (batch_seq_len < 1024) {
      block_dim = (batch_seq_len + 31) >> 5;
      block_dim *= 32;
    }

    t5_ker_correlation_softmax_encself<T>
        <<<dim3(batch_size, head_num * batch_seq_len), block_dim, 0, stream>>>(
            correlation, src_padding_mask, batch_seq_len, pos_emb);
  }

  template void t5_ker_correlation_softmax_encself_launcher<float>(
      int batch_size, int batch_seq_len, int head_num, hipStream_t stream,
      float* correlation, const int* src_padding_mask, const float *pos_emb);

  template void t5_ker_correlation_softmax_encself_launcher<__half>(
      int batch_size, int batch_seq_len, int head_num, hipStream_t stream,
      __half* correlation, const int* src_padding_mask, const __half *pos_emb);

  template void t5_ker_norm_layer_launcher<float>(int token_num, int hidden_size,
                                             hipStream_t stream, float* matrix, float* out,
                                             const float* scale,
                                             const float* bias,
                                             int max_thread_per_block);

  template void t5_ker_norm_layer_launcher<__half>(
      int token_num, int hidden_size, hipStream_t stream, __half* matrix, __half* out,
      const __half* scale, const __half* bias, int max_thread_per_block);


  /**
  @brief: t5_ker_correlation_softmax_decself
  query-key correlation softmax for decoder self attention

  @thread
  gridDim.x = batch_size * beam_size * head_num
  blockDim.x = first multiple of WARP_SIZE greater than cur_step + 1
  
  @param
  correlation: [batch_size, beam_size, head_num, cur_step + 1]
  */
  template <typename T>
  __global__ void t5_ker_correlation_softmax_decself(T* correlation, int step_num, const T *pos_emb) {
    int idx = blockIdx.x * step_num + threadIdx.x;
    // float val =
    //     threadIdx.x < step_num ? (float)correlation[idx] : CUDA_FLOAT_INF_NEG;

    float val;
    if (threadIdx.x < step_num) {
      // blockIdx.x = head_num + beam_size * 8 + batch_size * 8 * beam_size
      int j = threadIdx.x;
      int i = step_num - 1;
      int head_idx = blockIdx.x % 8;
      val = (float)correlation[idx];
      int bucket_index = get_bucket_num(i, j, false);
      val += (float)pos_emb[bucket_index * 8 + head_idx];
    } else val = CUDA_FLOAT_INF_NEG;
  
    float max_val = blockReduceMax(val);
    __shared__ float smax;
    if (threadIdx.x == 0) smax = max_val;
    __syncthreads();

    val = threadIdx.x < step_num ? expf(val - smax) : 0;

    float rsum = blockReduceSum(val);
    __shared__ float ssum;
    if (threadIdx.x == 0) ssum = rsum;
    __syncthreads();

    if (threadIdx.x < step_num) correlation[idx] = (T)(val / ssum);
  }


  template <typename T>
  void t5_ker_correlation_softmax_decself_launcher(int batch_head_num, int step_num,
                                                hipStream_t stream,
                                                T* correlation, const T *pos_emb) {
    int block_dim = step_num;
    if (step_num < 1024) {
      block_dim = (step_num + 31) >> 5;
      block_dim *= 32;
    }
    t5_ker_correlation_softmax_decself<<<batch_head_num, block_dim, 0, stream>>>(
        correlation, step_num, pos_emb);
  }
  
  template void t5_ker_correlation_softmax_decself_launcher<float>(
      int batch_head_num, int step_num, hipStream_t stream, float* correlation, const float *pos_emb);
  
  template void t5_ker_correlation_softmax_decself_launcher<__half>(
      int batch_head_num, int step_num, hipStream_t stream, __half* correlation, const __half *pos_emb);
  
}  // namespace cuda
}  // namespace lightseq
