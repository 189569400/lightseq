#include "hip/hip_runtime.h"
#include <random>

#include "common.h"
#include "gptKernels_int8.h"
#include "transformerKernels.h"
/**
@file
Implemented the cuda kernel function and its launcher
that required by GPT model.
Currently, fp16 and fp32 versions are provided
*/
namespace lightseq {
namespace cuda {

template <typename T>
__global__ void ker_gpt_embedding_int8(const int8_t* token_emb,
                                       const T* pos_emb, const int* token_id,
                                       T* output, int* real_seq_len,
                                       int padding_id, int pos_offset,
                                       float dequant_scale) {
  int target_pos = blockIdx.x * gridDim.y + blockIdx.y;
  int tid = token_id[target_pos];
  if (tid == padding_id) {
    // for padding id
    output[target_pos * blockDim.x + threadIdx.x] = 0.f;
    return;
  }
  if (threadIdx.x == 0) {
    atomicAdd(real_seq_len + blockIdx.x, 1);
  }
  output[target_pos * blockDim.x + threadIdx.x] =
      T(token_emb[tid * blockDim.x + threadIdx.x]) * dequant_scale +
      pos_emb[(blockIdx.y + pos_offset) * blockDim.x + threadIdx.x];
}

/* fp16 version */
template <>
__global__ void ker_gpt_embedding_int8<__half>(
    const int8_t* token_emb, const __half* pos_emb, const int* token_id,
    __half* output, int* real_seq_len, int padding_id, int pos_offset,
    float dequant_scale) {
  int target_pos = blockIdx.x * gridDim.y + blockIdx.y;
  int tid = token_id[target_pos];
  half2* output_h = (half2*)output;

  if (tid == padding_id) {
    // for padding id
    output_h[target_pos * blockDim.x + threadIdx.x] = __float2half2_rn(0.f);
    return;
  }
  if (threadIdx.x == 0) {
    atomicAdd(real_seq_len + blockIdx.x, 1);
  }

  float2 te;
  char2 cte = ((const char2*)token_emb)[tid * blockDim.x + threadIdx.x];
  float2 pe = __half22float2(
      ((const half2*)
           pos_emb)[(blockIdx.y + pos_offset) * blockDim.x + threadIdx.x]);
  te.x = float(cte.x) * dequant_scale + pe.x;
  te.y = float(cte.y) * dequant_scale + pe.y;
  output_h[target_pos * blockDim.x + threadIdx.x] = __float22half2_rn(te);
}

template <typename T>
void ker_gpt_embedding_i8I_launcher(int batch_size, int batch_seq_len,
                                    int hidden_size, hipStream_t stream,
                                    const int8_t* token_emb, const T* pos_emb,
                                    const int* token_id, T* output,
                                    int* real_seq_len, int padding_id,
                                    int pos_offset, float dequant_scale) {
  ker_gpt_embedding_int8<T>
      <<<dim3(batch_size, batch_seq_len), hidden_size, 0, stream>>>(
          token_emb, pos_emb, token_id, output, real_seq_len, padding_id,
          pos_offset, dequant_scale);
}

template <>
void ker_gpt_embedding_i8I_launcher<__half>(
    int batch_size, int batch_seq_len, int hidden_size, hipStream_t stream,
    const int8_t* token_emb, const __half* pos_emb, const int* token_id,
    __half* output, int* real_seq_len, int padding_id, int pos_offset,
    float dequant_scale) {
  ker_gpt_embedding_int8<__half>
      <<<dim3(batch_size, batch_seq_len), hidden_size / 2, 0, stream>>>(
          token_emb, pos_emb, token_id, output, real_seq_len, padding_id,
          pos_offset, dequant_scale);
}

template void ker_gpt_embedding_i8I_launcher<float>(
    int batch_size, int batch_seq_len, int hidden_size, hipStream_t stream,
    const int8_t* token_emb, const float* pos_emb, const int* token_id,
    float* output, int* real_seq_len, int padding_id, int pos_offset,
    float dequant_scale);

template void ker_gpt_embedding_i8I_launcher<__half>(
    int batch_size, int batch_seq_len, int hidden_size, hipStream_t stream,
    const int8_t* token_emb, const __half* pos_emb, const int* token_id,
    __half* output, int* real_seq_len, int padding_id, int pos_offset,
    float dequant_scale);

__global__ void ker_ppl_i8I(const int8_t* logits, const int* input_ids,
                            const int* real_seq_len, float* ppl, int vocab_size,
                            float dequant_scale, bool in_col32) {
  int seq_len = real_seq_len[blockIdx.x];  // remove "eos"
  if (blockIdx.y >= seq_len - 1) {
    // will not contribute to ppl
    return;
  }

  int token_idx_in_batch = blockIdx.x * gridDim.y + blockIdx.y;
  int left_logit_idx = token_idx_in_batch * vocab_size + threadIdx.x;
  int right_logit_idx = (token_idx_in_batch + 1) * vocab_size;
  /*
  step 1. find max logit over the whole vocab
  */
  float max_logit = CUDA_FLOAT_INF_NEG;
  for (int idx = left_logit_idx; idx < right_logit_idx; idx += blockDim.x) {
    int logits_idx;
    if (in_col32) {
      int row_id = token_idx_in_batch;
      int col_id = idx - token_idx_in_batch * vocab_size;
      logits_idx = row_major2flat_col32(row_id, col_id, gridDim.x * gridDim.y,
                                        vocab_size);
    } else {
      logits_idx = idx;
    }
    max_logit = fmaxf(max_logit, (float)logits[logits_idx] * dequant_scale);
  }
  max_logit = blockReduceMax(max_logit);
  __shared__ float s_max_logit;
  if (threadIdx.x == 0) {
    s_max_logit = max_logit;
  }
  __syncthreads();

  /*
  step 2. compute the log probability for the given token,
  add it to the sequence's ppl
  */
  float sum_exp_logit = 0.f;
  for (int idx = left_logit_idx; idx < right_logit_idx; idx += blockDim.x) {
    int logits_idx;
    if (in_col32) {
      int row_id = token_idx_in_batch;
      int col_id = idx - token_idx_in_batch * vocab_size;
      logits_idx = row_major2flat_col32(row_id, col_id, gridDim.x * gridDim.y,
                                        vocab_size);
    } else {
      logits_idx = idx;
    }
    float lgt = fmaxf((float)logits[logits_idx] * dequant_scale - s_max_logit,
                      logit_thresh_min);
    sum_exp_logit += expf(lgt);
  }
  sum_exp_logit = blockReduceSum(sum_exp_logit);

  if (threadIdx.x == 0) {
    int token_id = input_ids[token_idx_in_batch + 1];
    int logits_idx;
    if (in_col32) {
      int row_id = token_idx_in_batch;
      int col_id = token_id;
      logits_idx = row_major2flat_col32(row_id, col_id, gridDim.x * gridDim.y,
                                        vocab_size);
    } else {
      logits_idx = token_idx_in_batch * vocab_size + token_id;
    }
    float log_prob = ((float)logits[logits_idx] * dequant_scale - s_max_logit -
                      logf(sum_exp_logit)) /
                     (float)(seq_len - 1);
    atomicAdd(ppl + blockIdx.x, -log_prob);
  }
}

void ker_ppl_i8I_launcher(int batch_size, int batch_seq_len,
                          int max_thread_per_block, hipStream_t stream,
                          const int8_t* logits, const int* input_ids,
                          const int* real_seq_len, float* ppl, int vocab_size,
                          float dequant_scale, bool in_col32) {
  ker_ppl_i8I<<<dim3(batch_size, batch_seq_len), max_thread_per_block, 0,
                stream>>>(logits, input_ids, real_seq_len, ppl, vocab_size,
                          dequant_scale, in_col32);
}

}  // namespace cuda
}  // namespace lightseq
