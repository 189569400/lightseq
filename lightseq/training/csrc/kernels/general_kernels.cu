#include "hip/hip_runtime.h"
#include "kernels.h"
#include <iostream>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

/**
@brief: fuse_transpose_bias
Calculate the sum of elements in each column of the matrix.

@thread
gridDim.x = ceil(cols / WARP_SIZE)
blockDim.x = WARP_SIZE
blockDim.y = WARP_SIZE

@param
inp: [rows, cols]
out: [cols]
rows: the number of rows in the matrix
cols: the number of cols in the matrix
*/
template <typename T>
__global__ void column_sum_reduce(const T *__restrict__ inp,
                                  T *__restrict__ out, int rows, int cols) {
  __shared__ float tile[WARP_SIZE][WARP_SIZE];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

  int idx = flat_2dim(blockIdx.x, threadIdx.x, WARP_SIZE);
  int y_stride = cols * WARP_SIZE;
  float localSum = 0;

  // Loop across matrix row
  // TODO: optimize to log complexity
  if (idx < cols) {
    int offset = flat_2dim(threadIdx.y, idx, cols);
    for (int r = threadIdx.y; r < rows; r += WARP_SIZE) {
      localSum += (float)inp[offset];
      offset += y_stride;
    }
  }

  // The sum of a row in tile is equal to the sum of a col in original matrix
  tile[threadIdx.x][threadIdx.y] = localSum;

  __syncthreads();

  // Sum the shared buffer.
  // The change of threadIdx.x is continuous
  float sum = tile[threadIdx.y][threadIdx.x];

  __syncthreads();

  // Calculate the sum of a row in tile
  for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_down(sum, i);

  if (threadIdx.x == 0) {
    int pos = flat_2dim(blockIdx.x, threadIdx.y, WARP_SIZE);
    if (pos < cols) out[pos] = sum;
  }
}

// [r, c] -> [c]
template <>
void launch_fuse_transpose_bias_kernel<float>(const float *inp, float *out,
                                              int rows, int cols,
                                              hipStream_t stream) {
  dim3 grid_dim((cols - 1) / WARP_SIZE + 1);
  dim3 block_dim(WARP_SIZE, WARP_SIZE);

  column_sum_reduce<float>
      <<<grid_dim, block_dim, 0, stream>>>(inp, out, rows, cols);
}

template <>
void launch_fuse_transpose_bias_kernel<__half>(const __half *inp, __half *out,
                                               int rows, int cols,
                                               hipStream_t stream) {
  dim3 grid_dim((cols - 1) / WARP_SIZE + 1);
  dim3 block_dim(WARP_SIZE, WARP_SIZE);

  column_sum_reduce<__half>
      <<<grid_dim, block_dim, 0, stream>>>(inp, out, rows, cols);
}

/**
@brief: fused_add2
Add two matrix inp1 and inp2 to out.

@thread
gridDim.x = batch_size * seq_len
blockDim.x = min(hidden_dim, MAX_THREADS)

@param
inp1: [batch_size, seq_len, hidden_dim]
inp2: [batch_size, seq_len, hidden_dim]
out: [batch_size, seq_len, hidden_dim]
batch_size: the size of the current batch
seq_len: the sequence length of the current batch
hidden_dim: dim of the hidden tensor
*/
template <typename T>
__global__ void fused_add2_kernel(T *out, const T *inp1, const T *inp2,
                                  int hidden_dim);

template <>
__global__ void fused_add2_kernel<float>(float *out, const float *inp1,
                                         const float *inp2, int hidden_dim) {
  int row_id = blockIdx.x;
  int offset = flat_2dim(row_id, 0, hidden_dim);

  const float4 *inp1_4 = reinterpret_cast<const float4 *>(inp1);
  const float4 *inp2_4 = reinterpret_cast<const float4 *>(inp2);
  float4 *out_4 = reinterpret_cast<float4 *>(out);
  float4 vinp1;
  float4 vinp2;
  float4 val;

  for (std::size_t i = threadIdx.x; i < hidden_dim; i += blockDim.x) {
    vinp1 = inp1_4[offset + i];
    vinp2 = inp2_4[offset + i];
    val.x = vinp1.x + vinp2.x;
    val.y = vinp1.y + vinp2.y;
    val.z = vinp1.z + vinp2.z;
    val.w = vinp1.w + vinp2.w;
    out_4[offset + i] = val;
  }
}

template <>
__global__ void fused_add2_kernel<__half>(__half *out, const __half *inp1,
                                          const __half *inp2, int hidden_dim) {
  int row_id = blockIdx.x;
  int offset = flat_2dim(row_id, 0, hidden_dim);

  const float4 *inp1_4 = reinterpret_cast<const float4 *>(inp1);
  const float4 *inp2_4 = reinterpret_cast<const float4 *>(inp2);
  float4 *out_4 = reinterpret_cast<float4 *>(out);
  float4 vinp1;
  float4 vinp2;
  float4 val;
  __half2 *h2_inp1 = reinterpret_cast<__half2 *>(&vinp1);
  __half2 *h2_inp2 = reinterpret_cast<__half2 *>(&vinp2);
  __half2 *h2_val = reinterpret_cast<__half2 *>(&val);

  for (std::size_t i = threadIdx.x; i < hidden_dim; i += blockDim.x) {
    vinp1 = inp1_4[offset + i];
    vinp2 = inp2_4[offset + i];
    h2_val[0] = __hadd2(h2_inp1[0], h2_inp2[0]);
    h2_val[1] = __hadd2(h2_inp1[1], h2_inp2[1]);
    h2_val[2] = __hadd2(h2_inp1[2], h2_inp2[2]);
    h2_val[3] = __hadd2(h2_inp1[3], h2_inp2[3]);
    out_4[offset + i] = val;
  }
}

//[b, s, h] -> [b, s, h]
template <>
void launch_fused_add2<float>(float *out, const float *inp1, const float *inp2,
                              int batch_size, int seq_len, int hidden_dim,
                              hipStream_t &stream) {
  hidden_dim >>= 2;

  dim3 grid_dim(batch_size * seq_len);
  dim3 block_dim(min(hidden_dim, MAX_THREADS));

  fused_add2_kernel<<<grid_dim, block_dim, 0, stream>>>(out, inp1, inp2,
                                                        hidden_dim);
}

template <>
void launch_fused_add2<__half>(__half *out, const __half *inp1,
                               const __half *inp2, int batch_size, int seq_len,
                               int hidden_dim, hipStream_t &stream) {
  hidden_dim >>= 3;

  dim3 grid_dim(batch_size * seq_len);
  dim3 block_dim(min(hidden_dim, MAX_THREADS));

  fused_add2_kernel<<<grid_dim, block_dim, 0, stream>>>(out, inp1, inp2,
                                                        hidden_dim);
}

template <typename T>
__global__ void kernel_concat3_dim1(const T *inp1, const T *inp2, T *output,
                                    int sz0, int sz2, int sz1_1, int sz1_2) {
  int nele = sz0 * sz2 * (sz1_1 + sz1_2);
  int idx = flat_2dim(blockIdx.x, threadIdx.x, blockDim.x);
  if (idx >= nele) {
    return;
  }
  float4 *dst_ptr = (float4 *)output + idx;
  int idx2 = idx % sz2;
  idx = idx / sz2;
  int idx1 = idx % (sz1_1 + sz1_2);
  int idx0 = idx / (sz1_1 + sz1_2);
  float4 *src_ptr = nullptr;
  int sz1 = 0;
  if (idx1 < sz1_1) {
    sz1 = sz1_1;
    src_ptr = (float4 *)inp1;
  } else {
    idx1 -= sz1_1;
    sz1 = sz1_2;
    src_ptr = (float4 *)inp2;
  }
  src_ptr += flat_3dim(idx0, idx1, idx2, sz1, sz2);
  dst_ptr[0] = src_ptr[0];
}

template <>
void launch_concat3_dim1<float>(const float *inp1, const float *inp2,
                                float *output, int sz0, int sz2, int sz1_1,
                                int sz1_2, hipStream_t stream) {
  sz2 >>= 2;
  int nele = sz0 * sz2 * (sz1_1 + sz1_2);
  int nblock = (nele + MAX_THREADS - 1) / MAX_THREADS;
  kernel_concat3_dim1<<<nblock, MAX_THREADS, 0, stream>>>(
      inp1, inp2, output, sz0, sz2, sz1_1, sz1_2);
}

template <>
void launch_concat3_dim1<__half>(const __half *inp1, const __half *inp2,
                                 __half *output, int sz0, int sz2, int sz1_1,
                                 int sz1_2, hipStream_t stream) {
  sz2 >>= 3;
  int nele = sz0 * sz2 * (sz1_1 + sz1_2);
  int nblock = (nele + MAX_THREADS - 1) / MAX_THREADS;
  kernel_concat3_dim1<<<nblock, MAX_THREADS, 0, stream>>>(
      inp1, inp2, output, sz0, sz2, sz1_1, sz1_2);
}

/**
@brief: ker_split_multilg_request
request = numpy.concatenate((src_lang_id, trg_lang_id, src_token_id), axis=1)

@thread
gridDim.x = (nele + MAX_THREADS - 1) / MAX_THREADS
blockDim.x = MAX_THREADS

@param
inp1: [batch_size, seq_len, hidden_dim]
inp2: [batch_size, seq_len, hidden_dim]
out: [batch_size, seq_len, hidden_dim]
batch_size: the size of the current batch
seq_len: the sequence length of the current batch
hidden_dim: dim of the hidden tensor
*/
__global__ void ker_split_multilg_request(const int *req, int *src_lang_id,
                                          int *trg_lang_id, int *src_token_id,
                                          int batch_size, int req_len) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < batch_size * req_len) {
    int value = req[idx];
    int seq_id = idx / req_len;
    int token_id = idx % req_len;

    if (token_id == 0) {
      src_lang_id[seq_id] = value;
    } else if (token_id == 1) {
      trg_lang_id[seq_id] = value;
    } else {
      int new_idx = flat_2dim(seq_id, token_id - 2, req_len - 2);
      src_token_id[new_idx] = value;
    }
  }
}

void launch_split_multilg_request(const int *req, int *src_lang_id,
                                  int *trg_lang_id, int *src_token_id,
                                  int batch_size, int req_len,
                                  hipStream_t &stream) {
  if (req_len < 3) {
    throw std::runtime_error("req_len should be greater than 2");
  }
  int nele = batch_size * req_len;
  int nblock = (nele + MAX_THREADS - 1) / MAX_THREADS;
  ker_split_multilg_request<<<nblock, MAX_THREADS, 0, stream>>>(
      req, src_lang_id, trg_lang_id, src_token_id, batch_size, req_len);
}

/**
@brief: ker_enc_emb
for encoder, look up token embedding, add position embedding

@thread
gridDim.x = batch_size
gridDim.y = batch_seq_len
blockDim.x = max_thread_per_block

@param
token_emb: [vocab_size, hidden_size]
pos_emb: [max_step, hidden_size]
token_id: input token id, [batch_size, batch_seq_len]
output: result, [batch_size, batch_seq_len, hidden_size]
padding_mask: record the padding token, [batch_size, batch_seq_len]
padding_id, the padding token id
*/
template <typename T>
__global__ void ker_enc_emb(const T *token_emb, const T *pos_emb,
                            const int *token_id, T *output, int *padding_mask,
                            int padding_id, const int hidden_size) {
  int target_pos = blockIdx.x * gridDim.y + blockIdx.y;
  int start = target_pos * hidden_size + threadIdx.x;
  int end = (target_pos + 1) * hidden_size;
  int tid = token_id[target_pos];
  if (tid == padding_id) {
    // for padding id
    if (threadIdx.x == 0) padding_mask[target_pos] = 1;
    for (uint i = start; i < end; i += blockDim.x) {
      // output[target_pos * blockDim.x + threadIdx.x] = 0.f;
      output[i] = 0.f;
    }
    return;
  }
  if (threadIdx.x == 0) {
    padding_mask[target_pos] = 0;
  }
  for (uint i = start; i < end; i += blockDim.x) {
    int offset = i - target_pos * hidden_size;
    output[i] = token_emb[tid * hidden_size + offset] +
                pos_emb[blockIdx.y * hidden_size + offset];
  }
}

template <>
__global__ void ker_enc_emb<__half>(const __half *token_emb,
                                    const __half *pos_emb, const int *token_id,
                                    __half *output, int *padding_mask,
                                    int padding_id,
                                    const int half_hidden_size) {
  int target_pos = blockIdx.x * gridDim.y + blockIdx.y;
  int start = target_pos * half_hidden_size + threadIdx.x;
  int end = (target_pos + 1) * half_hidden_size;
  int tid = token_id[target_pos];
  half2 *output_h = (half2 *)output;

  if (tid == padding_id) {
    // for padding id
    if (threadIdx.x == 0) padding_mask[target_pos] = 1;
    for (uint i = start; i < end; i += blockDim.x) {
      output_h[i] = __float2half2_rn(0.f);
    }
    return;
  }
  if (threadIdx.x == 0) {
    padding_mask[target_pos] = 0;
  }
  for (uint i = start; i < end; i += blockDim.x) {
    int offset = i - target_pos * half_hidden_size;
    float2 te = __half22float2(
        ((const half2 *)token_emb)[tid * half_hidden_size + offset]);
    float2 pe = __half22float2(
        ((const half2 *)pos_emb)[blockIdx.y * half_hidden_size + offset]);
    te.x += pe.x;
    te.y += pe.y;
    output_h[i] = __float22half2_rn(te);
  }
}

template <typename T>
void launch_enc_emb(int batch_size, int batch_seq_len, int hidden_size,
                    hipStream_t stream, const T *token_emb, const T *pos_emb,
                    const int *token_id, T *output, int *padding_mask,
                    int padding_id, int max_thread_per_block) {
  ker_enc_emb<T>
      <<<dim3(batch_size, batch_seq_len), max_thread_per_block, 0, stream>>>(
          token_emb, pos_emb, token_id, output, padding_mask, padding_id,
          hidden_size);
}

template <>
void launch_enc_emb<__half>(int batch_size, int batch_seq_len, int hidden_size,
                            hipStream_t stream, const __half *token_emb,
                            const __half *pos_emb, const int *token_id,
                            __half *output, int *padding_mask, int padding_id,
                            int max_thread_per_block) {
  ker_enc_emb<__half>
      <<<dim3(batch_size, batch_seq_len), max_thread_per_block, 0, stream>>>(
          token_emb, pos_emb, token_id, output, padding_mask, padding_id,
          hidden_size / 2);
}

template void launch_enc_emb<float>(int batch_size, int batch_seq_len,
                                    int hidden_size, hipStream_t stream,
                                    const float *token_emb,
                                    const float *pos_emb, const int *token_id,
                                    float *output, int *padding_mask,
                                    int padding_id, int max_thread_per_block);

template void launch_enc_emb<__half>(int batch_size, int batch_seq_len,
                                     int hidden_size, hipStream_t stream,
                                     const __half *token_emb,
                                     const __half *pos_emb, const int *token_id,
                                     __half *output, int *padding_mask,
                                     int padding_id, int max_thread_per_block);
