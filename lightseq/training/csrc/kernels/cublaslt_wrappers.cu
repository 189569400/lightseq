/* Copyright 2021 The LightSeq Team
*/
#include "cublaslt_wrappers.h"


template <typename T>
int cublas_lt_matmul(hipblasLtHandle_t handle, hipblasLtMatmulDesc_t matmulDesc,
                     hipblasLtMatrixLayout_t ADesc, hipblasLtMatrixLayout_t BDesc,
                     hipblasLtMatrixLayout_t CDesc, T *A, T *B, T *C, T *alpha,
                     T *beta, hipStream_t stream) {
  hipblasStatus_t status;
  status = hipblasLtMatmul(handle, matmulDesc, alpha, A, ADesc, B, BDesc, beta,
                          C, CDesc, C, CDesc, nullptr, nullptr, 0, stream);
  return status;
}
