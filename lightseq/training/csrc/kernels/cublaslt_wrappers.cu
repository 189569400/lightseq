#include "hip/hip_runtime.h"
/* Copyright 2021 The LightSeq Team
 */
#include "cublaslt_wrappers.h"

int cublas_lt_matmul(hipblasLtHandle_t handle, hipblasLtMatmulDesc_t matmulDesc,
                     hipblasLtMatrixLayout_t ADesc, hipblasLtMatrixLayout_t BDesc,
                     hipblasLtMatrixLayout_t CDesc, const float *A,
                     const float *B, float *C, float *alpha, float *beta,
                     hipStream_t stream) {
  hipblasStatus_t status;
  status = hipblasLtMatmul(handle, matmulDesc, alpha, A, ADesc, B, BDesc, beta,
                          C, CDesc, C, CDesc, nullptr, nullptr, 0, stream);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! kernel execution error. (error: %d) \n", (int)status);
    return EXIT_FAILURE;
  }
  return 0;
}

int cublas_lt_matmul(hipblasLtHandle_t handle, hipblasLtMatmulDesc_t matmulDesc,
                     hipblasLtMatrixLayout_t ADesc, hipblasLtMatrixLayout_t BDesc,
                     hipblasLtMatrixLayout_t CDesc, const __half *A,
                     const __half *B, __half *C, __half *alpha, __half *beta,
                     hipStream_t stream) {
  hipblasStatus_t status;
  status = hipblasLtMatmul(handle, matmulDesc, alpha, A, ADesc, B, BDesc, beta,
                          C, CDesc, C, CDesc, nullptr, nullptr, 0, stream);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! kernel execution error. (error: %d) \n", (int)status);
    return EXIT_FAILURE;
  }
  return 0;
}
