#include "hip/hip_runtime.h"
#include <chrono>
#include <ctime>

#include "int8_kernels.h"

__host__ __device__ int8_t float2int(float x, float scale, float clip_max) {
  if (x > clip_max) x = clip_max;
  if (x < -clip_max) x = -clip_max;
  int8_t y = int8_t(x / clip_max * scale);
  return y;
}

template <typename T>
__global__ void quantize_tensor_kernel(const T *input, int8_t *output,
                                       int total_count, float scale,
                                       float clip_max);

template <>
__global__ void quantize_tensor_kernel<float>(const float *input,
                                              int8_t *output, int total_count,
                                              float scale, float clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i * 4 >= total_count) return;

  const float4 *input4 = reinterpret_cast<const float4 *>(input);
  int32_t *output4 = reinterpret_cast<int32_t *>(output);
  float4 inp4 = input4[i];
  int32_t out4;
  int8_t *out1 = reinterpret_cast<int8_t *>(&out4);
  out1[0] = float2int(inp4.x, scale, clip_max);
  out1[1] = float2int(inp4.y, scale, clip_max);
  out1[2] = float2int(inp4.z, scale, clip_max);
  out1[3] = float2int(inp4.w, scale, clip_max);
  output4[i] = out4;
}

template <>
__global__ void quantize_tensor_kernel<__half>(const __half *input,
                                               int8_t *output, int total_count,
                                               float scale, float clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i * 8 >= total_count) return;

  const float4 *input4 = reinterpret_cast<const float4 *>(input);
  int64_t *output4 = reinterpret_cast<int64_t *>(output);
  float4 inp4 = input4[i];
  int64_t out8;
  __half *inp_h = reinterpret_cast<__half *>(&inp4);
  int8_t *out1 = reinterpret_cast<int8_t *>(&out8);
#pragma unroll
  for (uint j = 0; j < 8; ++j) {
    out1[j] = float2int(__half2float(inp_h[j]), scale, clip_max);
  }
  output4[i] = out8;
}

template <>
void launch_quantize_tensor<float>(const float *input, int8_t *output,
                                   int total_count, float scale, float clip_max,
                                   hipStream_t &stream) {
  int grid_dim = total_count >> 12;
  quantize_tensor_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      input, output, total_count, scale, clip_max);
}

template <>
void launch_quantize_tensor<__half>(const __half *input, int8_t *output,
                                    int total_count, float scale,
                                    float clip_max, hipStream_t &stream) {
  int grid_dim = total_count >> 13;
  quantize_tensor_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      input, output, total_count, scale, clip_max);
}

template <typename T>
__global__ void dequantize_tensor_kernel(const int32_t *input, T *output,
                                         int total_count, float scale,
                                         float clip_max);

template <>
__global__ void dequantize_tensor_kernel<float>(const int32_t *input,
                                                float *output, int total_count,
                                                float scale, float clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i * 4 >= total_count) return;

  const int4 *input4 = reinterpret_cast<const int4 *>(input);
  float4 *output4 = reinterpret_cast<float4 *>(output);
  int4 inp4 = input4[i];
  float4 out4;
  out4.x = float(inp4.x) / scale * clip_max;
  out4.y = float(inp4.y) / scale * clip_max;
  out4.z = float(inp4.z) / scale * clip_max;
  out4.w = float(inp4.w) / scale * clip_max;
  output4[i] = out4;
}

template <>
__global__ void dequantize_tensor_kernel<__half>(const int32_t *input,
                                                 __half *output,
                                                 int total_count, float scale,
                                                 float clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i * 4 >= total_count) return;

  const int4 *input4 = reinterpret_cast<const int4 *>(input);
  float2 *output4 = reinterpret_cast<float2 *>(output);
  int4 inp4 = input4[i];
  float2 out4;
  int32_t *inp1 = reinterpret_cast<int32_t *>(&inp4);
  __half *out1 = reinterpret_cast<__half *>(&out4);
#pragma unroll
  for (uint j = 0; j < 4; ++j) {
    out1[j] = __float2half(float(inp1[j]) / scale * clip_max);
  }
  output4[i] = out4;
}

template <>
void launch_dequantize_tensor<float>(const int32_t *input, float *output,
                                     int total_count, float scale,
                                     float clip_max, hipStream_t &stream) {
  int grid_dim = total_count >> 12;
  dequantize_tensor_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      input, output, total_count, scale, clip_max);
}

template <>
void launch_dequantize_tensor<__half>(const int32_t *input, __half *output,
                                      int total_count, float scale,
                                      float clip_max, hipStream_t &stream) {
  int grid_dim = total_count >> 12;
  dequantize_tensor_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      input, output, total_count, scale, clip_max);
}
