#include "hip/hip_runtime.h"
#include <chrono>
#include <ctime>

#include "block_reduce.h"
#include "int8_kernels.h"

const float LN_EPSILON = 1e-8f;

__forceinline__ __host__ __device__ int8_t float2int8(float x,
                                                      float scale_div_clipmax,
                                                      float clip_max) {
  x = x > clip_max ? clip_max : (x < -clip_max ? -clip_max : x);
  return int8_t(x * scale_div_clipmax);
}

template <typename T>
__global__ void quantize_tensor_kernel(const T *input, int8_t *output,
                                       int total_count, float scale,
                                       float clip_max);

template <>
__global__ void quantize_tensor_kernel<float>(const float *input,
                                              int8_t *output, int total_count,
                                              float scale_div_clipmax,
                                              float clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i * 4 >= total_count) return;

  const float4 *input4 = reinterpret_cast<const float4 *>(input);
  int32_t *output4 = reinterpret_cast<int32_t *>(output);
  float4 inp4 = input4[i];
  int32_t out4;
  int8_t *out1 = reinterpret_cast<int8_t *>(&out4);
  out1[0] = float2int8(inp4.x, scale_div_clipmax, clip_max);
  out1[1] = float2int8(inp4.y, scale_div_clipmax, clip_max);
  out1[2] = float2int8(inp4.z, scale_div_clipmax, clip_max);
  out1[3] = float2int8(inp4.w, scale_div_clipmax, clip_max);
  output4[i] = out4;
}

template <>
__global__ void quantize_tensor_kernel<__half>(const __half *input,
                                               int8_t *output, int total_count,
                                               float scale_div_clipmax,
                                               float clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i * 8 >= total_count) return;

  const float4 *input4 = reinterpret_cast<const float4 *>(input);
  int64_t *output4 = reinterpret_cast<int64_t *>(output);
  float4 inp4 = input4[i];
  int64_t out8;
  __half *inp_h = reinterpret_cast<__half *>(&inp4);
  int8_t *out1 = reinterpret_cast<int8_t *>(&out8);
#pragma unroll
  for (uint j = 0; j < 8; ++j) {
    out1[j] = float2int8(__half2float(inp_h[j]), scale_div_clipmax, clip_max);
  }
  output4[i] = out8;
}

template <>
void launch_quantize_tensor<float>(const float *input, int8_t *output,
                                   int total_count, float scale, float clip_max,
                                   hipStream_t &stream) {
  int grid_dim = total_count >> 12;
  quantize_tensor_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      input, output, total_count, scale / clip_max, clip_max);
}

template <>
void launch_quantize_tensor<__half>(const __half *input, int8_t *output,
                                    int total_count, float scale,
                                    float clip_max, hipStream_t &stream) {
  int grid_dim = total_count >> 13;
  quantize_tensor_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      input, output, total_count, scale / clip_max, clip_max);
}

template <typename T>
__global__ void dequantize_tensor_kernel(const int32_t *input, T *output,
                                         int total_count,
                                         float scale_div_clipmax,
                                         float clip_max);

template <>
__global__ void dequantize_tensor_kernel<float>(const int32_t *input,
                                                float *output, int total_count,
                                                float scale_div_clipmax,
                                                float clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i * 4 >= total_count) return;

  const int4 *input4 = reinterpret_cast<const int4 *>(input);
  float4 *output4 = reinterpret_cast<float4 *>(output);
  int4 inp4 = input4[i];
  float4 out4;
  out4.x = float(inp4.x) / scale_div_clipmax;
  out4.y = float(inp4.y) / scale_div_clipmax;
  out4.z = float(inp4.z) / scale_div_clipmax;
  out4.w = float(inp4.w) / scale_div_clipmax;
  output4[i] = out4;
}

template <>
__global__ void dequantize_tensor_kernel<__half>(const int32_t *input,
                                                 __half *output,
                                                 int total_count,
                                                 float scale_div_clipmax,
                                                 float clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i * 8 >= total_count) return;

  const long4 *input4 = reinterpret_cast<const long4 *>(input);
  float4 *output4 = reinterpret_cast<float4 *>(output);
  long4 inp4 = input4[i];
  float4 out4;
  int32_t *inp1 = reinterpret_cast<int32_t *>(&inp4);
  __half *out1 = reinterpret_cast<__half *>(&out4);
#pragma unroll
  for (uint j = 0; j < 8; ++j) {
    out1[j] = __float2half(float(inp1[j]) / scale_div_clipmax);
  }
  output4[i] = out4;
}

template <>
void launch_dequantize_tensor<float>(const int32_t *input, float *output,
                                     int total_count, float scale,
                                     float clip_max, hipStream_t &stream) {
  int grid_dim = total_count >> 12;
  dequantize_tensor_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      input, output, total_count, scale / clip_max, clip_max);
}

template <>
void launch_dequantize_tensor<__half>(const int32_t *input, __half *output,
                                      int total_count, float scale,
                                      float clip_max, hipStream_t &stream) {
  int grid_dim = total_count >> 13;
  dequantize_tensor_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      input, output, total_count, scale / clip_max, clip_max);
}

void trans_weight(int8_t *input, int8_t *output, int m, int n,
                  hipStream_t &stream) {
  hipblasLtHandle_t handle;
  hipblasLtCreate(&handle);
  hipblasLtOrder_t order_COL32 = CUBLASLT_ORDER_COL32;
  hipblasLtMatrixLayout_t input_desc, output_desc;
  hipblasLtMatrixTransformDesc_t transform_desc;
  hipblasOperation_t opTrans = HIPBLAS_OP_T;
  int ld_input = n, ld_output = 32 * m;
  float alpha = 1.0, beta = 0.0;
  CHECK_GPU_ERROR(
      hipblasLtMatrixLayoutCreate(&input_desc, HIP_R_8I, n, m, ld_input));
  CHECK_GPU_ERROR(
      hipblasLtMatrixLayoutCreate(&output_desc, HIP_R_8I, m, n, ld_output));
  CHECK_GPU_ERROR(hipblasLtMatrixLayoutSetAttribute(
      output_desc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32,
      sizeof(order_COL32)));

  CHECK_GPU_ERROR(
      hipblasLtMatrixTransformDescCreate(&transform_desc, HIP_R_32F));
  CHECK_GPU_ERROR(hipblasLtMatrixTransformDescSetAttribute(
      transform_desc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTrans,
      sizeof(opTrans)));
  CHECK_GPU_ERROR(hipblasLtMatrixTransform(handle, transform_desc, &alpha, input,
                                          input_desc, &beta, NULL, NULL, output,
                                          output_desc, stream));
  CHECK_GPU_ERROR(hipblasLtMatrixLayoutDestroy(input_desc));
  CHECK_GPU_ERROR(hipblasLtMatrixLayoutDestroy(output_desc));
  CHECK_GPU_ERROR(hipblasLtMatrixTransformDescDestroy(transform_desc));
}

template <typename T>
void quant_trans_weight(const T *input, int8_t *output, int m, int n,
                        float scale, float clip_max) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  int8_t *buffer = cuda_malloc<int8_t>(m * n);
  launch_quantize_tensor(input, buffer, m * n, scale, clip_max, stream);
  trans_weight(buffer, output, m, n, stream);
  cuda_free(buffer);
}

template void quant_trans_weight<float>(const float *input, int8_t *output,
                                        int m, int n, float scale,
                                        float clip_max);
template void quant_trans_weight<__half>(const __half *input, int8_t *output,
                                         int m, int n, float scale,
                                         float clip_max);

/**
@brief: ker_layer_norm
Standard layer normalization.
It will not only output the layer norm result,
  but also outputs variance.
  may also output means, depends on whether
  the means argument is nullptr

@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
ln_res: [batch_size* seq_len, hidden_size], ln result.
vars: [batch_size* seq_len], variance per token
means: [batch_size* seq_len], means per token, can be nullput
inp: [batch_size * seq_len, hidden_size], ln input.
scale: [hidden_size], ln scale
bias: [hidden_size], ln bias
*/
template <typename T>
__global__ void ker_layer_norm_int8O(int8_t *ln_res, T *vars, T *means,
                                     const T *inp, const T *scale,
                                     const T *bias, int hidden_size,
                                     float quant_scale, float clip_max) {
  // step 0. compute local sum
  float l_sum = 0;
  float l_square_sum = 0;
  const float4 *inp_f4 = (const float4 *)inp + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val = inp_f4[idx];
    l_sum += val.x + val.y + val.z + val.w;
    l_square_sum +=
        val.x * val.x + val.y * val.y + val.z * val.z + val.w * val.w;
  }

  // step 1. compute reduce sum
  float mean_dim = float(hidden_size) * 4.f;
  float reduce_val[2] = {l_sum, l_square_sum};
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_mean, s_var;
  if (threadIdx.x == 0) {
    s_mean = reduce_val[0] / mean_dim;
    if (means != nullptr) {
      means[blockIdx.x] = s_mean;
    }
    s_var = reduce_val[1] / mean_dim - s_mean * s_mean + LN_EPSILON;
    vars[blockIdx.x] = s_var;
    s_var = rsqrtf(s_var);
  }
  __syncthreads();

  // step 2. layer norm result
  char4 *output_i4 = (char4 *)ln_res + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 vscale = __ldg((const float4 *)scale + idx);
    float4 vbias = __ldg((const float4 *)bias + idx);
    float4 val = inp_f4[idx];
    char4 val_i4;
    val.x = (val.x - s_mean) * s_var * vscale.x + vbias.x;
    val.y = (val.y - s_mean) * s_var * vscale.y + vbias.y;
    val.z = (val.z - s_mean) * s_var * vscale.z + vbias.z;
    val.w = (val.w - s_mean) * s_var * vscale.w + vbias.w;
    val_i4.x = float2int8(val.x, quant_scale / clip_max, clip_max);
    val_i4.y = float2int8(val.y, quant_scale / clip_max, clip_max);
    val_i4.z = float2int8(val.z, quant_scale / clip_max, clip_max);
    val_i4.w = float2int8(val.w, quant_scale / clip_max, clip_max);
    output_i4[idx] = val_i4;
  }
}

template <>
__global__ void ker_layer_norm_int8O<__half>(int8_t *ln_res, __half *vars,
                                             __half *means, const __half *inp,
                                             const __half *scale,
                                             const __half *bias,
                                             int hidden_size, float quant_scale,
                                             float clip_max) {
  // step 0. compute local sum
  float l_sum = 0;
  float l_square_sum = 0;
  const float4 *inp_f4 = (const float4 *)inp + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val_f4 = inp_f4[idx];
    __half2 *val_h2 = (__half2 *)(&val_f4);
#pragma unroll
    for (int i = 0; i < 4; i++) {
      float2 val_f2 = __half22float2(val_h2[i]);
      l_sum += val_f2.x + val_f2.y;
      l_square_sum += val_f2.x * val_f2.x + val_f2.y * val_f2.y;
    }
  }

  // step 1. compute reduce sum
  float mean_dim = float(hidden_size) * 8.f;
  float reduce_val[2] = {l_sum, l_square_sum};
  blockReduce<ReduceType::kSum, 2>(reduce_val);
  __shared__ float s_mean, s_var;
  if (threadIdx.x == 0) {
    s_mean = reduce_val[0] / mean_dim;
    if (means != nullptr) {
      means[blockIdx.x] = s_mean;
    }
    s_var = reduce_val[1] / mean_dim - s_mean * s_mean + LN_EPSILON;
    vars[blockIdx.x] = s_var;
    s_var = rsqrtf(s_var);
  }
  __syncthreads();

  // step 2. layer norm result
  int64_t *output_i4 = (int64_t *)ln_res + blockIdx.x * hidden_size;
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    // load scale, bias, input
    float4 scale_f4 = __ldg((const float4 *)scale + idx);
    __half2 *scale_h2 = (__half2 *)(&scale_f4);
    float4 bias_f4 = __ldg((const float4 *)bias + idx);
    __half2 *bias_h2 = (__half2 *)(&bias_f4);
    float4 val_f4 = inp_f4[idx];
    __half2 *val_h2 = (__half2 *)(&val_f4);
    int64_t out8;
    int8_t *out1 = reinterpret_cast<int8_t *>(&out8);

#pragma unroll
    for (int i = 0; i < 4; i++) {
      float2 scale_f2 = __half22float2(scale_h2[i]);
      float2 bias_f2 = __half22float2(bias_h2[i]);
      float2 val_f2 = __half22float2(val_h2[i]);
      val_f2.x = (val_f2.x - s_mean) * s_var * scale_f2.x + bias_f2.x;
      val_f2.y = (val_f2.y - s_mean) * s_var * scale_f2.y + bias_f2.y;
      out1[i * 2] = float2int8(val_f2.x, quant_scale / clip_max, clip_max);
      out1[i * 2 + 1] = float2int8(val_f2.y, quant_scale / clip_max, clip_max);
    }
    output_i4[idx] = out8;
  }
}

template <>
void launch_layer_norm_int8O<float>(int8_t *ln_res, float *vars, float *means,
                                    const float *inp, const float *scale,
                                    const float *bias, int batch_size,
                                    int hidden_dim, float quant_scale,
                                    float clip_max, hipStream_t stream) {
  if (hidden_dim % 4 != 0) {
    throw std::runtime_error("violate hidden_dim % 4 = 0");
  }
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  dim3 grid_dim(batch_size);
  dim3 block_dim(nthread);

  ker_layer_norm_int8O<float><<<grid_dim, block_dim, 0, stream>>>(
      ln_res, vars, means, inp, scale, bias, hidden_dim, quant_scale, clip_max);
}

template <>
void launch_layer_norm_int8O<__half>(int8_t *ln_res, __half *vars,
                                     __half *means, const __half *inp,
                                     const __half *scale, const __half *bias,
                                     int batch_size, int hidden_dim,
                                     float quant_scale, float clip_max,
                                     hipStream_t stream) {
  if (hidden_dim % 8 != 0) {
    throw std::runtime_error("violate hidden_dim % 8 = 0");
  }
  hidden_dim >>= 3;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  dim3 grid_dim(batch_size);
  dim3 block_dim(nthread);

  ker_layer_norm_int8O<__half><<<grid_dim, block_dim, 0, stream>>>(
      ln_res, vars, means, inp, scale, bias, hidden_dim, quant_scale, clip_max);
}
