#include "hip/hip_runtime.h"
#include "transformer.h"

/**
@file
Example of how to run transformer inference using our implementation.
*/

int main(int argc, char *argv[]) {
  std::string model_weights_path = argv[1];
  int max_batch_size = 8;
  //   std::string res = tw_.initializing(model_weights_path);
  //   if (!res.empty()) {
  //     std::cout << res << std::endl;
  //     return 0;
  //   }
  auto model = lightseq::cuda::Transformer(model_weights_path, max_batch_size);
  /*
    step3. instantiate encoder and decoder, init the gpu memory buffer.
      using thrust vector to avoid manage gpu memory by hand
  */
  // instantiate encoder
  //   int max_batch_size = 8;
  //   thrust::device_vector<int> d_input_ =
  //       std::vector<int>(max_batch_size * tw_._max_step, 0);
  //   thrust::device_vector<int> d_padding_mask_ =
  //       std::vector<int>(max_batch_size * tw_._max_step, 0);
  //   thrust::device_vector<int> d_encoder_output_ =
  //       std::vector<int>(max_batch_size * tw_._max_step * tw_._hidden_size,
  //       0);
  //   thrust::device_vector<int> d_output_ =
  //       std::vector<int>(max_batch_size * tw_._max_step, 0);
  //   std::shared_ptr<lightseq::cuda::Encoder<OPTYPE>> encoder_ =
  //       std::make_shared<lightseq::cuda::Encoder<OPTYPE>>(
  //           max_batch_size,
  //           reinterpret_cast<int
  //           *>(thrust::raw_pointer_cast(d_input_.data())),
  //           reinterpret_cast<int *>(
  //               thrust::raw_pointer_cast(d_padding_mask_.data())),
  //           reinterpret_cast<optraits::DataType *>(
  //               thrust::raw_pointer_cast(d_encoder_output_.data())),
  //           tw_, stream_, hd_);
  //   res = encoder_->check();
  //   if (!res.empty()) {
  //     std::cout << res << std::endl;
  //     return 1;
  //   }
  // instantiate decoder
  //   std::shared_ptr<lightseq::cuda::Decoder<OPTYPE>> decoder_ =
  //       std::make_shared<lightseq::cuda::Decoder<OPTYPE>>(
  //           max_batch_size,
  //           reinterpret_cast<int *>(
  //               thrust::raw_pointer_cast(d_padding_mask_.data())),
  //           reinterpret_cast<optraits::DataType *>(
  //               thrust::raw_pointer_cast(d_encoder_output_.data())),
  //           reinterpret_cast<int
  //           *>(thrust::raw_pointer_cast(d_output_.data())), tw_, stream_,
  //           hd_, false, reinterpret_cast<int
  //           *>(thrust::raw_pointer_cast(d_input_.data())));
  //   res = decoder_->check();
  //   if (!res.empty()) {
  //     std::cout << res << std::endl;
  //     return 1;
  //   }
  // init gpu memory buffer
  //   long buf_bytesize = std::max(encoder_->compute_buffer_bytesize(),
  //                                decoder_->compute_buffer_bytesize());
  //   thrust::device_vector<int> d_buf_ =
  //       std::vector<int>(buf_bytesize / sizeof(int), 0);
  //   // encoder and decoder use the same buffer to save gpu memory useage
  //   encoder_->init_buffer(
  //       reinterpret_cast<void *>(thrust::raw_pointer_cast(d_buf_.data())));
  //   decoder_->init_buffer(
  //       reinterpret_cast<void *>(thrust::raw_pointer_cast(d_buf_.data())));
  //   hipStreamSynchronize(stream_);

  /* ---step4. read input token ids from file--- */
  int batch_size = 1;
  int batch_seq_len = 14;
  std::vector<int> host_input = {0,     100, 657, 14,    1816, 6, 53,
                                 50264, 473, 45,  50264, 162,  4, 2};
  // the first line of input file should
  // be two integers: batch_size and batch_seq_len.
  // followed by batch_size lines of
  // batch_seq_len integers, e.g.
  // 2 3
  // 666 666 666
  // 666 666 666
  //   std::string input_file_name = argv[2];
  //   lightseq::cuda::read_batch_tokenids_from_file(input_file_name,
  //   batch_size,
  //                                                 batch_seq_len, host_input);

  int *d_input;
  lightseq::cuda::CHECK_GPU_ERROR(
      hipMalloc(&d_input, sizeof(int) * batch_size * batch_seq_len));
  lightseq::cuda::CHECK_GPU_ERROR(hipMemcpy(
      d_input, host_input.data(), sizeof(int) * batch_size * batch_seq_len,
      hipMemcpyHostToDevice));
  /* ---step5. infer and log--- */
  for (int i = 0; i < 10; i++) {
    auto start = std::chrono::high_resolution_clock::now();
    // copy inputs from cpu memory to gpu memory
    // encoder_->run_one_infer(batch_size, batch_seq_len);
    // decoder_->run_one_infer(batch_size, batch_seq_len);
    model.infer(d_input, batch_size, batch_seq_len);
    lightseq::cuda::print_time_duration(start, "one infer time", 0);
    // for (int ii = 0; ii < batch_size; ii++) {
    //   lightseq::cuda::print_vec(
    //       d_output_.data() + ii * (decoder_->_cur_step + 1), "finial res",
    //       decoder_->_cur_step + 1);
    // }
  }
  return 0;
}
